#include "hip/hip_runtime.h"
// This program executes a typical convolutional layer in regular CNNs.Neuron sparsity(zero ratio) is 50% and Weight sparsity is 70%.
#include <iostream>
#include "CNNConvLayer.h"
using namespace std;

// This is the CPU version, please don't modify it
void convLayerCPU()
{
	// declarations for bunch of indexing parameters
	int fn, sli, fmy, fmx, y, x;
	int ifmy, ifmx, ofmy, ofmx;
	int filtIdx, inNeuIdx, outNeuIdx, outIdx;
	int filtVol  = FMDEPTH  * FILTSIZE * FILTSIZE;
	int fmArea   = FMSIZE   * FMSIZE;
	int filtArea = FILTSIZE * FILTSIZE;
	int outArea  = FMSIZE/3 * FMSIZE/3;
	int sum;
	// Convolution
	for(fn = 0; fn < FILTNUM; fn++){
		for(fmy = 0; fmy < FMSIZE; fmy += STRIDE){
			for(fmx = 0; fmx < FMSIZE; fmx += STRIDE){
				sum = 0;
				for(sli = 0; sli < FMDEPTH; sli++){
					for(y = 0; y < FILTSIZE; y++){
						for(x = 0; x < FILTSIZE; x++){
							ifmy = fmy - FILTSIZE / 2 + y;
							ifmx = fmx - FILTSIZE / 2 + x;
							filtIdx = fn*filtVol + sli*filtArea + y*FILTSIZE + x;
							inNeuIdx = sli*fmArea + ifmy*FMSIZE + ifmx;
							if(ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE)
								sum += filt[filtIdx] * inNeu[inNeuIdx];
						}
					}
				}
				// Activation - ReLU
				outNeuIdx = fn*fmArea + fmy*FMSIZE + fmx;
				if(sum <= 0)
					outNeu[outNeuIdx] = 0;
				else
					outNeu[outNeuIdx] = sum;
			}
		}
	}

	// Max Pooling with Window Size 3x3 and stride 3
	int max, tmpVal;
	for(sli = 0; sli < FILTNUM; sli++){
		for(fmy = 0; fmy < FMSIZE/3 ; fmy += 1){
			for(fmx = 0; fmx < FMSIZE/3 ; fmx += 1){
				outNeuIdx = sli*fmArea + fmy*3*FMSIZE + fmx*3;
				max = outNeu[outNeuIdx];
				for(y = 0; y < 3; y++){
					for(x = 0; x < 3; x++){
						ofmy = fmy*3 + y;
						ofmx = fmx*3 + x;
						outNeuIdx = sli*fmArea + ofmy*FMSIZE + ofmx;
						tmpVal = outNeu[outNeuIdx];	
						if(tmpVal > max)
							max = tmpVal;
					}
				}
				outIdx = sli*outArea + fmy*FMSIZE/3 + fmx;
				outCPU[outIdx] = max;
			}
		}
	}
}

/***	Implement your CUDA Kernel here	***/
__global__
void convLayerGPU()
{
}
/***	Implement your CUDA Kernel here	***/

int main()
{
	//variables setting and loading input data
	timespec time_begin, time_end; 
	int convLayerCPUExecTime, convLayerGPUExecTime;
	init();
	initCoo();

	//Convolution by CPU                                                
	clock_gettime(CLOCK_REALTIME, &time_begin);
	convLayerCPU();
	clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerCPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "CPU time for executing a typical convolutional layer = "  <<  ((float)convLayerCPUExecTime)/1000 << "ms" << endl;

  
	//Convolution by GPU   
	clock_gettime(CLOCK_REALTIME, &time_begin);
	/***	Lunch your CUDA Kernel here	***/
	convLayerGPU<<<1,1>>>(); // Lunch the kernel
	hipDeviceSynchronize(); // Do synchronization before clock_gettime()
	/***	Lunch your CUDA Kernel here	***/
	clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerGPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "GPU time for executing a typical convolutional layer = "  << ((float)convLayerGPUExecTime)/1000 << "ms" << endl;

	
	//check the anser from CPU and from GPU
	if(checker()){
		cout << "Congratulations! You pass the check." << endl;
		cout << "Speedup: " << (float)convLayerCPUExecTime / convLayerGPUExecTime << endl;
	}
	else
		cout << "Sorry! Your result is wrong." << endl;

	//release memory space
	ending();
	
	return 0;
}
