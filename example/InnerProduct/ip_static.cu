// This program will demo how to use CUDA to accelerate inner-product

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
using namespace std;

#define VECNUM 50000
#define VECLEN 1000
#define HALFNUM 25000000
int *inputA, *inputB;
int *devInputA, *devInputB, *devOut;
int *outCPU, *outGPU;

void init()
{
	delete outGPU;
	int i, j, idx;

	inputA = new int[VECNUM * VECLEN];
	inputB = new int[VECNUM * VECLEN];


	for(i = 0; i < VECNUM; i++){
		for(j = 0; j < VECLEN; j++){
			idx = i*VECLEN + j;
			if(idx<HALFNUM){
				if(idx%2==0){ //if(idx=even number) =>set value=1 
					inputA[idx] =1;
					inputB[idx] =1;
				}
				else{         //if(idx=odd number) =>set value=0
					inputA[idx] =0;
					inputB[idx] =0;
				}
			}
			else{
				inputA[idx] =3;
				inputB[idx] =3;
			}
			
			
		}
	}

	outCPU = new int[VECNUM]();
	outGPU = new int[VECNUM]();
	
}

void initGPU()
{
	int inputSize = sizeof(int)*VECNUM*VECLEN;

	hipMalloc(&devInputA, inputSize);
	hipMalloc(&devInputB, inputSize);
	hipMalloc(&devOut, sizeof(int)*VECNUM);

	hipMemcpy(devInputA, inputA, inputSize, hipMemcpyHostToDevice);
	hipMemcpy(devInputB, inputB, inputSize, hipMemcpyHostToDevice);
}

__global__ 
void innerProductGPU(int *A, int *B, int *out)
{
	int y = blockIdx.x;
	int x = threadIdx.x;
	__shared__ int tmp[VECLEN];

	int idx = y * VECLEN + x;
	tmp[x] = A[idx] * B[idx];
	
	__syncthreads();

	if(x == 0){
		int i, sum = 0;
		for(i = 0; i < VECLEN; i++)
			sum += tmp[i];
		out[y] = sum;
	}
}

void innerProductCPU()
{
	int i, j, acc, idx;

	for(i = 0; i < VECNUM; i++){
		acc = 0;
		for(j = 0; j < VECLEN; j++){
			idx = i*VECLEN + j;
			acc += inputA[idx] * inputB[idx];
		}
		outCPU[i] = acc;
	}
}

bool checker(){
	int i;

	for(i = 0; i < VECNUM; i++){ 
		if(outCPU[i] != outGPU[i]){
			cout << "The element: " << i << " is wrong!\n";
			cout << "outCPU[" << i << "] = " << outCPU[i] << endl;
			cout << "outGPU[" << i << "] = " << outGPU[i] << endl;
			return false;
		}
	}

	return true;
}

int timespec_diff_us(timespec& t1, timespec& t2)
{                                                                                
  return (t2.tv_sec - t1.tv_sec) * 1e6 + (t2.tv_nsec - t1.tv_nsec) / 1e3;        
} 

int main()
{
	int outSize = sizeof(int)*VECNUM;
	init();
	initGPU();
	timespec time_begin, time_end;    
	
    clock_gettime(CLOCK_REALTIME, &time_begin);
	innerProductCPU();
    clock_gettime(CLOCK_REALTIME, &time_end);
	// cout << "CPU time for executing inner-product = " << timespec_diff_us(time_begin, time_end) / 1000 << "ms" << endl;

	// GPU static version
	dim3 threadsPerBlock(VECLEN);
	dim3 numBlocks(VECNUM);
	
    clock_gettime(CLOCK_REALTIME, &time_begin);
	innerProductGPU<<<numBlocks, threadsPerBlock>>>(devInputA, devInputB, devOut);
	hipDeviceSynchronize();
    clock_gettime(CLOCK_REALTIME, &time_end);
	cout << "GPU time for executing static inner-product = " << timespec_diff_us(time_begin, time_end)  << "us" << endl;

	//data copy from GPU to CPU
	hipMemcpy(outGPU, devOut, outSize, hipMemcpyDeviceToHost);
	
	//check
	if(checker())
		cout << "Congratulations! You pass the check." << endl;
	else
		cout << "Sorry! Your result is wrong." << endl;
	
	//releas space
	hipFree(&devInputA);
	hipFree(&devInputB);
	hipFree(&devOut);
	delete outGPU;

	return 0;
}
