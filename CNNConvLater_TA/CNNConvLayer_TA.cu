#include "hip/hip_runtime.h"
// This program executes a typical convolutional layer in regular CNNs
#include <iostream>
#include <stdio.h>
#include "CNNConvLayer_TA.h"
using namespace std;
// This is the CPU version, please don't modify it
void convLayerCPU()
{
	// declarations for bunch of indexing parameters
	int fn, sli, fmy, fmx, y, x;
	int sum, ifmy, ifmx, ofmy, ofmx;
	int filtIdx, inNeuIdx, outNeuIdx, outIdx;
	int filtVol = FMDEPTH * FILTSIZE * FILTSIZE;
	int filtArea = FILTSIZE * FILTSIZE;
	int fmArea = FMSIZE *FMSIZE;
	int outArea = FMSIZE/2 * FMSIZE/2;

	// Convolution
	for(fn = 0; fn < FILTNUM; fn++){
		for(fmy = 0; fmy < FMSIZE; fmy += STRIDE){
			for(fmx = 0; fmx < FMSIZE; fmx += STRIDE){
				sum = 0;
				for(sli = 0; sli < FMDEPTH; sli++){
					for(y = 0; y < FILTSIZE; y++){
						for(x = 0; x < FILTSIZE; x++){
							ifmy = fmy - FILTSIZE / 2 + y;
							ifmx = fmx - FILTSIZE / 2 + x;
							filtIdx = fn*filtVol + sli*filtArea + y*FILTSIZE + x;
							inNeuIdx = sli*fmArea + ifmy*FMSIZE + ifmx;
							if(ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE)
								sum += filt[filtIdx] * inNeu[inNeuIdx];
						}
					}
				}
				// Activation - ReLU
				outNeuIdx = fn*fmArea + fmy*FMSIZE + fmx;
				if(sum <= 0)
					outNeu[outNeuIdx] = 0;
				else
					outNeu[outNeuIdx] = sum;
			}
		}
	}

	// Max Pooling with Window Size 2x2
	int max, tmpVal;
	for(sli = 0; sli < FILTNUM; sli++){
		for(fmy = 0; fmy < FMSIZE/2 ; fmy += 1){
			for(fmx = 0; fmx < FMSIZE/2 ; fmx += 1){
				outNeuIdx = sli*fmArea + fmy*2*FMSIZE + fmx*2;
				max = outNeu[outNeuIdx];
				for(y = 0; y < 2; y++){
					for(x = 0; x < 2; x++){
						ofmy = fmy*2 + y;
						ofmx = fmx*2 + x;
						outNeuIdx = sli*fmArea + ofmy*FMSIZE + ofmx;
						tmpVal = outNeu[outNeuIdx];	
						if(tmpVal > max)
							max = tmpVal;
					}
				}
				outIdx = sli*outArea + fmy*FMSIZE/2 + fmx;
				outCPU[outIdx] = max;
			}
		}
	}
}

int *devInputA, *devInputB;
int *convOut;
int *devOut;

void initGPU()
{   	
	hipMalloc(&devInputA, sizeof(int)* FMDEPTH * FMSIZE * FMSIZE );
	hipMalloc(&devInputB, sizeof(int)*FILTSIZE*FILTSIZE*FMDEPTH*FILTNUM );
	hipMalloc(&convOut, sizeof(int)* FMSIZE * FMSIZE *FILTNUM  );
	hipMalloc(&devOut, sizeof(int)* FMSIZE * FMSIZE *FILTNUM /4  );

	hipMemcpy(devInputA, inNeu, sizeof(int)* FMDEPTH * FMSIZE * FMSIZE, hipMemcpyHostToDevice);
	hipMemcpy(devInputB, filt, sizeof(int)*FILTSIZE*FILTSIZE*FMDEPTH*FILTNUM, hipMemcpyHostToDevice);
	
}

__global__
void convLayerGPU(int *inNeu , int *filt , int *outNeu)
{
    int fn, sli, fmy, fmx, y, x ,p ;
	int ifmy, ifmx;
	int filtIdx, inNeuIdx, outNeuIdx;
	int filtVol = FMDEPTH * FILTSIZE * FILTSIZE;
	int filtArea = FILTSIZE * FILTSIZE;
	int fmArea = FMSIZE *FMSIZE;
    int total_sum;
	__shared__ int sum[FMDEPTH];
	
	sli=threadIdx.x;
		
	for(fn = 0; fn < FILTNUM; fn++){
		for(fmy = 0; fmy < FMSIZE; fmy += STRIDE){
			for(fmx = 0; fmx < FMSIZE; fmx += STRIDE){
				sum[sli] = 0;
				for(y = 0; y < FILTSIZE; y++){
					for(x = 0; x < FILTSIZE; x++){
						ifmy = fmy - FILTSIZE / 2 + y;
						ifmx = fmx - FILTSIZE / 2 + x;
						filtIdx = fn*filtVol + sli*filtArea + y*FILTSIZE + x;
						inNeuIdx = sli*fmArea + ifmy*FMSIZE + ifmx;
						
						if(ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE){
							sum[sli] += filt[filtIdx] * inNeu[inNeuIdx];
							if(threadIdx.x==0){
								
							}
						}
							
					}
				}
				__syncthreads();
				
				if(threadIdx.x==0){
					total_sum =0;
					for(p=0;p<FMDEPTH;p++)
						total_sum=total_sum+sum[p];
					
					outNeuIdx = fn*fmArea + fmy*FMSIZE + fmx;
					if(total_sum <= 0)
						outNeu[outNeuIdx] = 0;
					else
						outNeu[outNeuIdx] = total_sum;
				}
				__syncthreads();				
			}
		}
	}
				
}

__global__
void poolLayerGPU(int *inNeu , int *outNeu)
{	
	int Tx = threadIdx.x;
	int Ty = threadIdx.y;
    int block = blockIdx.x;
	int inNeu_ID = block*FMSIZE*FMSIZE + Ty*FMSIZE*2 + Tx*2 ;
	int Max = inNeu[inNeu_ID];

	if( Max < inNeu[inNeu_ID+1] )
		Max = inNeu[inNeu_ID+1];
	else
		Max = Max;
	 __syncthreads();
	if( Max < inNeu[inNeu_ID+FMSIZE] )
		Max = inNeu[inNeu_ID+FMSIZE];
	else
		Max = Max;
	 __syncthreads();
	if( Max < inNeu[inNeu_ID+FMSIZE+1] )
		Max = inNeu[inNeu_ID+FMSIZE+1];
	else
		Max = Max;

	 __syncthreads();
	outNeu[block*FMSIZE/2*FMSIZE/2 + Ty*FMSIZE/2 + Tx] = Max;
	
}


int main()
{
	int convLayerCPUExecTime, convLayerGPUExecTime;
	init();
	initGPU();

	timespec time_begin, time_end;                                                 
	clock_gettime(CLOCK_REALTIME, &time_begin);
	convLayerCPU();
	clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerCPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "CPU time for executing a typical convolutional layer = " <<  convLayerCPUExecTime / 1000 << "ms" << endl;

  	clock_gettime(CLOCK_REALTIME, &time_begin);
	convLayerGPU<<<1,FMDEPTH>>>(devInputA,devInputB,convOut); 
	hipDeviceSynchronize(); 
	dim3 threadsPerBlock_pool(FMSIZE/2,FMSIZE/2);
	dim3 numBlocks_pool(FILTNUM,1);
	poolLayerGPU<<<numBlocks_pool,threadsPerBlock_pool>>>(convOut,devOut);
	hipDeviceSynchronize(); 
	
	hipMemcpy(outGPU, devOut , sizeof(int) * FMSIZE * FMSIZE *FILTNUM  /4, hipMemcpyDeviceToHost);
	
	hipFree(&devInputA);
	hipFree(&devInputB);
	hipFree(&convOut);
	hipFree(&devOut);
	
	clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerGPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "GPU time for executing a typical convolutional layer = " 
			 << convLayerGPUExecTime / 1000 << "ms" << endl;

	if(checker()){
		cout << "Congratulations! You pass the check." << endl;
		cout << "Speedup: " << (float)convLayerCPUExecTime / convLayerGPUExecTime << endl;
	}
	else
		cout << "Sorry! Your result is wrong." << endl;

	ending();
	
	return 0;
}
